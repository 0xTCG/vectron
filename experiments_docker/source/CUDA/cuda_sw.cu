#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <utility>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iomanip> 
#include "utils.h"


constexpr int SIZE = 512; 
int QUANTITY = 0;
constexpr int CUDA_XBLOCK_SIZE = 256; 

using dp_mat = float[SIZE + 1][SIZE + 1];

__global__
void align(float *scores, dp_mat *matrices, dp_mat *matrices_left, dp_mat *matrices_top, char *sequences, size_t size) {
    const int t = threadIdx.x + blockIdx.x * blockDim.x;

    if (t >= QUANTITY)
        return;

    const float gap_o = -4.0f;
    const float mismatch = -4.0f;
    const float match = 2.0f;
    const float gap_e = -2.0f;
    const float ambig = -3.0f;

    matrices[t][0][0] = 0.0f;
    matrices_left[t][0][0] = -10000.0f;
    matrices_top[t][0][0] = -10000.0f;
    for (int i = 1; i < size + 1; ++i) {
        matrices[t][0][i] = gap_o + gap_e * i;
        matrices[t][i][0] = gap_o + gap_e * i;
        matrices_left[t][0][i] = gap_o + gap_e * i;
        matrices_left[t][i][0] = -10000.0f;
        matrices_top[t][0][i] = -10000.0f;
        matrices_top[t][i][0] = gap_o + gap_e * i;
    }

    float target_value;
    float max_value = 0.0f;
    for (int i = 1; i < size + 1; ++i) {
        for (int j = 1; j < size + 1; ++j) {
            if (j - i <= -105 || j - i >= 105) {
                matrices[t][i][j] = -10000.0f;
                matrices_left[t][i][j] = -10000.0f;
                matrices_top[t][i][j] = -10000.0f;   
            }
            else {
                float diagonal_value = matrices[t][i - 1][j - 1];
                if (sequences[t * size * 2 + i - 1] == 'N' || sequences[t * size * 2 + j - 1 + size] == 'N') {
                    diagonal_value += ambig;
                } else {
                    diagonal_value += (sequences[t * size * 2 + i - 1] == sequences[t * size * 2 + j - 1 + size] ? match : mismatch);
                }            
                float top_value =  (matrices[t][i - 1][j] + gap_o + gap_e > matrices_top[t][i - 1][j] + gap_e ? matrices[t][i - 1][j] + gap_o + gap_e : matrices_top[t][i - 1][j] + gap_e);
                float left_value = (matrices[t][i][j - 1] + gap_o + gap_e > matrices_left[t][i][j - 1] + gap_e ? matrices[t][i][j - 1] + gap_o + gap_e : matrices_left[t][i][j - 1] + gap_e);
                float temp = top_value - ((top_value - left_value) * (top_value - left_value < 0));
                target_value = diagonal_value - ((diagonal_value - temp) * (diagonal_value - temp < 0));
                matrices[t][i][j] = target_value;
                matrices_left[t][i][j] = left_value;
                matrices_top[t][i][j] = top_value;        
                if(target_value > max_value)
                    max_value = target_value;
            } 
        }
    }
    scores[t] = target_value;
}

void sw_cuda_alpern(std::vector<std::pair<std::string, std::string>> const sequences) {
    int const num_blocks = QUANTITY / CUDA_XBLOCK_SIZE;

    std::vector<float> scores(QUANTITY);

    dp_mat *dev_matrices;
    dp_mat *dev_matrices_top;
    dp_mat *dev_matrices_left;
    char *dev_input;
    float *dev_output;
    int64_t matrices_size = QUANTITY * sizeof(dp_mat);
    int64_t output_size = QUANTITY * sizeof(float);
    int64_t input_size = QUANTITY * SIZE * 2;
    auto const start_time = std::chrono::steady_clock::now();
    hipMalloc((void **)&dev_output, output_size);
    hipMalloc((void **)&dev_matrices, matrices_size);
    hipMalloc((void **)&dev_matrices_top, matrices_size);
    hipMalloc((void **)&dev_matrices_left, matrices_size);
    hipMalloc((void **)&dev_input, input_size);

    const char *sequences_bytes = to_byte_arr(sequences);

    hipMalloc((void **)&dev_input, input_size);
    hipError_t mallocError = hipGetLastError();
    if (mallocError != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(mallocError) << std::endl;
        return; 
    }
    hipMemcpy(dev_input, sequences_bytes, input_size, hipMemcpyHostToDevice);
    auto const start_time_kernel = std::chrono::steady_clock::now();
    align<<<num_blocks, CUDA_XBLOCK_SIZE>>>(dev_output, dev_matrices, dev_matrices_left, dev_matrices_top, dev_input, SIZE);
    auto const end_time_kernel = std::chrono::steady_clock::now();    
    hipMemcpy(scores.data(), dev_output, output_size, hipMemcpyDeviceToHost);
    
    hipFree(dev_input);
    hipFree(dev_matrices);
    hipFree(dev_matrices_left);
    hipFree(dev_matrices_top);
    hipFree(dev_output);
    //for (auto e : scores) {
    //    std::cout << e << "\n";
    //}
    // Simulating the score printing CPU load
    volatile float dummy;
    for (auto e : scores) {
        dummy = e;
    }
    auto const end_time = std::chrono::steady_clock::now();
    std::cout << std::fixed << std::setprecision(2) 
          << std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count() / 1e6 
          << std::endl;
}

std::vector<std::string> read_sequences_from_file(const std::string &filename) {
    std::ifstream file(filename);
    std::vector<std::string> sequences;
    std::string line;
    while (std::getline(file, line)) {
        sequences.push_back(line);
    }
    return sequences;
}

std::vector<std::pair<std::string, std::string>> pair_sequences(const std::vector<std::string> &targets,
                                                               const std::vector<std::string> &queries) {
    std::vector<std::pair<std::string, std::string>> pairs;
    for (size_t i = 0; i < targets.size(); ++i) {
        pairs.emplace_back(targets[i], queries[i]);
    }
    return pairs;
}

int main(int argc, char* argv[]) {
    std::string target_file = argv[argc - 1];
    std::string query_file = argv[argc - 2];

    std::vector<std::string> target_sequences = read_sequences_from_file(target_file);
    std::vector<std::string> query_sequences = read_sequences_from_file(query_file);
    QUANTITY = target_sequences.size();
    if (target_sequences.size() != query_sequences.size()) {
        std::cerr << "Error: Number of target sequences does not match number of query sequences." << std::endl;
        std::cout << "Target Size: " << target_sequences.size() << ", Query Size: " << query_sequences.size() << "\n";
        return 1;
    }

    std::vector<std::pair<std::string, std::string>> paired_sequences = pair_sequences(target_sequences, query_sequences);

    sw_cuda_alpern(paired_sequences);

    return 0;
}
